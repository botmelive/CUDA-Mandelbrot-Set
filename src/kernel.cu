#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define TX 32
#define TY 32
#define MAX_ITER 6

typedef double Float;

__device__ unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__ float shiftrange(float c, float d, float t)
{
   return c + (float)(d - c) * t;
}

/*
x2:= 0
y2:= 0

while (x2 + y2 ≤ 4 and iteration < max_iteration) do
    y:= 2 * x * y + y0
    x:= x2 - y2 + x0
    x2:= x * x
    y2:= y * y
    iteration:= iteration + 1
*/

__device__ int mandelbrot(Float x0, Float y0, int MAX)
{
   /*https://en.wikipedia.org/wiki/Plotting_algorithms_for_the_Mandelbrot_set*/
   Float x = 0.0f;
   Float y = 0.0f;
   Float x2 = 0.0f;
   Float y2 = 0.0f;
   int iteration = 0;

   while (x * x + y * y <= 4 && iteration < MAX)
   {
      y = (x + x) * y + y0;
      x = x2 - y2 + x0;
      x2 = x * x;
      y2 = y * y;
      iteration++;
   }

   return iteration;
}

__device__ void HSV2RGB(float H, float S, float V, int &R, int &G, int &B)
{
   float nNormalizedH = (float)H * 0.003921569F; // / 255.0F
   float nNormalizedS = (float)S * 0.003921569F;
   float nNormalizedV = (float)V * 0.003921569F;
   float nR;
   float nG;
   float nB;
   if (nNormalizedS == 0.0F)
   {
      nR = nG = nB = nNormalizedV;
   }
   else
   {
      if (nNormalizedH == 1.0F)
         nNormalizedH = 0.0F;
      else
         nNormalizedH = nNormalizedH * 6.0F; // / 0.1667F
   }
   float nI = floorf(nNormalizedH);
   float nF = nNormalizedH - nI;
   float nM = nNormalizedV * (1.0F - nNormalizedS);
   float nN = nNormalizedV * (1.0F - nNormalizedS * nF);
   float nK = nNormalizedV * (1.0F - nNormalizedS * (1.0F - nF));
   if (nI == 0.0F)
   {
      nR = nNormalizedV;
      nG = nK;
      nB = nM;
   }
   else if (nI == 1.0F)
   {
      nR = nN;
      nG = nNormalizedV;
      nB = nM;
   }
   else if (nI == 2.0F)
   {
      nR = nM;
      nG = nNormalizedV;
      nB = nK;
   }
   else if (nI == 3.0F)
   {
      nR = nM;
      nG = nN;
      nB = nNormalizedV;
   }
   else if (nI == 4.0F)
   {
      nR = nK;
      nG = nM;
      nB = nNormalizedV;
   }
   else if (nI == 5.0F)
   {
      nR = nNormalizedV;
      nG = nM;
      nB = nN;
   }
   R = (int)(nR * 255.0F);
   G = (int)(nG * 255.0F);
   B = (int)(nB * 255.0F);
}

__global__ void mandelbrotKernel(uchar4 *d_out, int w, int h, int MAX, Bounds bounds)
{
   const int c = blockIdx.x * blockDim.x + threadIdx.x;
   const int r = blockIdx.y * blockDim.y + threadIdx.y;

   if ((c >= w) || (r >= h))
      return; // Check if within image bounds

   const int i = c + r * w; // 1D indexing

   Float x = (Float)c / w;
   Float y = (Float)r / h;

   Float x_scaled = shiftrange(bounds.B1.x, bounds.B2.x, x);
   Float y_scaled = shiftrange(bounds.B1.y, bounds.B2.y, y);

   int iter = mandelbrot(x_scaled, y_scaled, MAX);

   int color = 0;
   int R = 0;
   int G = 0;
   int B = 0;
   if (iter < MAX)
   {
      int H = 255 * ((float)iter / MAX);
      int S = 255;
      int L = iter < MAX ? 255 : 0;

      HSV2RGB(H, S, L, R, G, B);
   }

   d_out[i].x = R;
   d_out[i].y = G;
   d_out[i].z = B;
   d_out[i].w = 255;
}

void kernelLauncher(uchar4 *d_out, int w, int h, int color, Bounds bounds)
{
   const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);
   const dim3 blockSize(TX, TY);
   mandelbrotKernel<<<gridSize, blockSize>>>(d_out, w, h, color, bounds);
}